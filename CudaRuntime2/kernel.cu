﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <cstdlib>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void kernel1() { printf("Hello, world!\n"); }

__global__ void kernel2() {
  int k = blockIdx.x * blockDim.x + threadIdx.x;
  printf("I am from %d block, %d thread (global index: %d)\n", blockIdx.x,
         threadIdx.x, k);
}

__global__ void kernel3(int* a, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    a[i] += i;
  }
}

int main() {
  int n = 10;
  const int block_size = 256;
  const int num_block = (n + block_size - 1) / block_size;

  kernel1<<<2, 2>>>();

  kernel2<<<2, block_size>>>();

  int* a = new int[n];
  int* a_gpu;
  for (int i = 0; i < n; i++) {
    a[i] = 0;
  }
  hipMalloc((void**)&a_gpu, n * sizeof(int));

  hipMemcpy(a_gpu, a, n * sizeof(int), hipMemcpyHostToDevice);

  kernel3<<<num_block, block_size>>>(a_gpu, n);

  hipMemcpy(a, a_gpu, n * sizeof(int), hipMemcpyDeviceToHost);

  for (int i = 0; i < n; i++) {
    printf("%d\n", a[i]);
  }
  delete[] a;
  hipFree(a_gpu);

  hipDeviceSynchronize();
  return 0;
}
